#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "mj_common.h"
#include "../../operator.h"
#include "../../utils/cuda_utils.h"

extern "C" void dense_layer_consume_tuple_cuda(ModeljoinState *state, int logical_layer, Buffer data, int row) 
{
    long node_in = ((long*)data[Node_in_off])[row];
    long node = ((long*)data[Node_off])[row];
    
    /* We later want to compute x*A, but sgemm only offers A*x. 
    Therefore we save the transposed matrix to compute x*A = A_t * x_t 
    Consequently, the element (node_in, node) is saved at (node, node_in), 
    which translates to node + node_in * dim */
    cuda_check_error(hipMemcpy(&(state->W_i[logical_layer][node + node_in * state->layer_dims[logical_layer]]), &(((float*)data[W_i_off])[row]),
        sizeof(float), hipMemcpyHostToDevice));
    cuda_check_error(hipMemcpy(&(state->b_i[logical_layer][node]), &(((float*)data[b_i_off])[row]),
        sizeof(float), hipMemcpyHostToDevice));
}

extern "C" void dense_layer_finish_cuda(void *o, int layer) 
{
    Operator *op = (Operator*)o;
    ModeljoinState *state = (ModeljoinState*) op->state;
    int cell_dim = state->layer_dims[layer];
    int i;

    float *new_bias;
    cuda_check_error(hipMalloc(&new_bias, cell_dim * op->vectorsize * sizeof(float)));

    for (i = 0; i < op->vectorsize; i++) {
        cuda_check_error(hipMemcpy(&(new_bias[i * cell_dim]), state->b_i[layer], cell_dim * sizeof(float), hipMemcpyDeviceToDevice));
    }

    cuda_check_error(hipFree(state->b_i[layer]));

    state->b_i[layer] = new_bias;
}

extern "C" float* dense_layer_forward_matrix_memcpy_loading_cuda(void *o, int layer, float *intermediate, int *int_rows, int *int_cols)
{
    Operator *op = (Operator*)o;
    ModeljoinState *state = (ModeljoinState*) op->state;
    /* cols of the transposed matrix is equal to the vector length */
    int cols = intermediate ? *int_rows : state->num_in_cols;
    /* rows is the layer's dimension and determines the output vector size of the layer */
    int rows = state->layer_dims[layer];
    int vectorsize = op->vectorsize;
    const float alpha = 1;
    const float beta = 1;
    const float gamma = 0;
    float *mat = state->W_i[layer];
    float *bias = state->b_i[layer];
    float *x;
    float *result;
    int i;
    bool allocated = false;
    hipblasHandle_t handle = *((hipblasHandle_t*)cublas_handle);

    cuda_check_error(hipMalloc(&result, rows * op->vectorsize * sizeof(float)));

    if (!intermediate) {
        /* Input layer */
        float *transposed;
        cuda_check_error(hipMalloc(&transposed, cols * vectorsize * sizeof(float)));
        cuda_check_error(hipMalloc(&x, cols * vectorsize * sizeof(float)));
        allocated = true;
        /* TODO: type */
        for (i = 0; i < cols; i++) {
            cuda_check_error(hipMemcpy(&(x[i * vectorsize]), ((float*)op->data[state->arg_col_map[i]]), 
                vectorsize * sizeof(float), hipMemcpyHostToDevice));   
        }
        /* Transpose */
        cublas_check_error(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, cols, vectorsize, &alpha, x, vectorsize, &gamma, transposed, cols, transposed, cols));
        cuda_check_error(hipFree(x));
        x = transposed;
    } else {
        x = intermediate;
    }
    cuda_check_error(hipMemcpy(result, bias, vectorsize * state->layer_dims[layer] * sizeof(float), hipMemcpyHostToDevice));
    cublas_check_error(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rows, vectorsize, cols, &alpha, mat, rows, x, cols, &beta, result, rows));
    if (allocated) cuda_check_error(hipFree(x));

    *int_rows = state->layer_dims[layer];
    *int_cols = vectorsize;
    return result;
}